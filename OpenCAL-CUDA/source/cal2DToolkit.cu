// (C) Copyright University of Calabria and others.
// All rights reserved. This program and the accompanying materials
// are made available under the terms of the GNU Lesser General Public License
// (LGPL) version 2.1 which accompanies this distribution, and is available at
// http://www.gnu.org/licenses/lgpl-2.1.html
//
// This library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
// Lesser General Public License for more details.

#include ".\..\include\cal2DToolkit.cuh"
#include <stdio.h>
#include ".\..\include\cal2DBufferIO.cuh"
#include ".\..\include\cal2DBuffer.cuh"
#include <iostream>
using namespace std;

struct CudaCALModel2D *copy_model;


struct CudaCALModel2D* calCudaAlloc(){
	struct CudaCALModel2D* device_object;
	hipMalloc((void**)&device_object, sizeof(CudaCALModel2D));
	return device_object;
}

struct CudaCALModel2D* calCudaHostAlloc(struct CudaCALModel2D *model){
	struct CudaCALModel2D* device_object;
	hipHostAlloc((void**)&device_object, sizeof(CudaCALModel2D), hipHostMallocDefault);

	hipMalloc((void**)&device_object->i,model->sizeof_X*sizeof(int));
	hipMalloc((void**)&device_object->j,model->sizeof_X*sizeof(int));

	if(model->OPTIMIZATION == CAL_OPT_ACTIVE_CELLS){
		hipMalloc((void**)&device_object->activecell_flags,model->rows*model->columns*sizeof(CALbyte));
		hipMalloc((void**)&device_object->activecell_index,model->rows*model->columns*sizeof(CALint));
	}

	if(model->sizeof_pQb_array > 0){
		hipMalloc((void**)&device_object->pQb_array_current,model->sizeof_pQb_array*model->rows*model->columns*sizeof(CALbyte));
		hipMalloc((void**)&device_object->pQb_array_next,model->sizeof_pQb_array*model->rows*model->columns*sizeof(CALbyte));
	}
	if(model->sizeof_pQi_array > 0){
		hipMalloc((void**)&device_object->pQi_array_current,model->sizeof_pQi_array*model->rows*model->columns*sizeof(CALint));
		hipMalloc((void**)&device_object->pQi_array_next,model->sizeof_pQi_array*model->rows*model->columns*sizeof(CALint));
	}
	if(model->sizeof_pQr_array > 0){
		hipMalloc((void**)&device_object->pQr_array_current,model->sizeof_pQr_array*model->rows*model->columns*sizeof(CALreal));
		hipMalloc((void**)&device_object->pQr_array_next,model->sizeof_pQr_array*model->rows*model->columns*sizeof(CALreal));
	}
	return device_object;
}

struct CudaCALModel2D* calCudaAllocatorModel(struct CudaCALModel2D *model){

	hipHostMalloc((void**)&copy_model, sizeof(struct CudaCALModel2D), hipHostMallocPortable);

	memcpy(copy_model,model,sizeof(struct CudaCALModel2D));

	hipMalloc((void**)&copy_model->i,model->sizeof_X*sizeof(int));
	hipMalloc((void**)&copy_model->j,model->sizeof_X*sizeof(int));

	if(model->OPTIMIZATION == CAL_OPT_ACTIVE_CELLS){
		hipMalloc((void**)&copy_model->activecell_flags,model->rows*model->columns*sizeof(CALbyte));
		hipMalloc((void**)&copy_model->activecell_index,model->rows*model->columns*sizeof(CALint));
		hipMalloc((void**)&copy_model->array_of_index_result, model->rows*model->columns*sizeof(CALint));
	}

	if(model->sizeof_pQb_array > 0){
		hipMalloc((void**)&copy_model->pQb_array_current,model->sizeof_pQb_array*model->rows*model->columns*sizeof(CALbyte));
		hipMalloc((void**)&copy_model->pQb_array_next,model->sizeof_pQb_array*model->rows*model->columns*sizeof(CALbyte));
	}
	if(model->sizeof_pQi_array > 0){
		hipMalloc((void**)&copy_model->pQi_array_current,model->sizeof_pQi_array*model->rows*model->columns*sizeof(CALint));
		hipMalloc((void**)&copy_model->pQi_array_next,model->sizeof_pQi_array*model->rows*model->columns*sizeof(CALint));
	}
	if(model->sizeof_pQr_array > 0){
		hipMalloc((void**)&copy_model->pQr_array_current,model->sizeof_pQr_array*model->rows*model->columns*sizeof(CALreal));
		hipMalloc((void**)&copy_model->pQr_array_next,model->sizeof_pQr_array*model->rows*model->columns*sizeof(CALreal));
	}

	return copy_model;
}

void calCudaFinalizeModel(){

	hipFree(copy_model->i);
	hipFree(copy_model->j);

	if(copy_model->OPTIMIZATION == CAL_OPT_ACTIVE_CELLS){
		hipFree(copy_model->activecell_flags);
		hipFree(copy_model->activecell_index);
		hipFree(copy_model->array_of_index_result);
	}
	hipFree(copy_model->pQb_array_current);
	hipFree(copy_model->pQb_array_next);
	hipFree(copy_model->pQi_array_current);
	hipFree(copy_model->pQi_array_next);
	hipFree(copy_model->pQr_array_current);
	hipFree(copy_model->pQr_array_next); 
	hipHostFree(copy_model);
}

void calCudaFreeModel2D(struct CudaCALModel2D *copy_model){

	hipFree(copy_model->i);
	hipFree(copy_model->j);

	if(copy_model->OPTIMIZATION == CAL_OPT_ACTIVE_CELLS){
		hipFree(copy_model->activecell_flags);
		hipFree(copy_model->activecell_index);
		hipFree(copy_model->array_of_index_result);
	}
	hipFree(copy_model->pQb_array_current);
	hipFree(copy_model->pQb_array_next);
	hipFree(copy_model->pQi_array_current);
	hipFree(copy_model->pQi_array_next);
	hipFree(copy_model->pQr_array_current);
	hipFree(copy_model->pQr_array_next); 
	hipHostFree(copy_model);
}

__device__
	CALint calCudaGetIndex(CudaCALModel2D* ca2D){

		if(ca2D->OPTIMIZATION == CAL_OPT_ACTIVE_CELLS){
			return ca2D->array_of_index_result[calCudaGetSimpleOffset()];
		}else{
			return calCudaGetSimpleOffset();
		}
}
__device__
	CALint calCudaGetSimpleOffset(){

		CALint i = blockIdx.x*blockDim.x + threadIdx.x;
		CALint j = blockIdx.y*blockDim.y + threadIdx.y;

		return i + j*blockDim.x*gridDim.x;
}

__device__ CALint calCudaGetIndexRow(CudaCALModel2D* model, CALint offset){
	return offset / (model->columns);
}

__device__ CALint calCudaGetIndexColumn(CudaCALModel2D* model, CALint offset){
	return offset % (model->columns);
}


CALbyte calInitializeInGPU2D(struct CudaCALModel2D* model, struct CudaCALModel2D *d_model){

	CALbyte result = CAL_TRUE;

	calCudaAllocatorModel(model);

	hipMemcpy(copy_model->i,model->i, sizeof(CALint)*model->sizeof_X, hipMemcpyHostToDevice);
	hipMemcpy(copy_model->j,model->j, sizeof(CALint)*model->sizeof_X, hipMemcpyHostToDevice);

	if(model->OPTIMIZATION == CAL_OPT_ACTIVE_CELLS){
		hipMemcpy(copy_model->activecell_flags,model->activecell_flags, sizeof(CALbyte)*model->rows*model->columns, hipMemcpyHostToDevice);
		hipMemcpy(copy_model->activecell_index,model->activecell_index, sizeof(CALint)*model->rows*model->columns, hipMemcpyHostToDevice);
		hipMemcpy(copy_model->array_of_index_result,model->array_of_index_result, sizeof(CALint)*model->rows*model->columns, hipMemcpyHostToDevice);
	}

	if(model->sizeof_pQb_array > 0){
		hipMemcpy(copy_model->pQb_array_current,model->pQb_array_current, model->sizeof_pQb_array*model->rows*model->columns*sizeof(CALbyte), hipMemcpyHostToDevice);
		hipMemcpy(copy_model->pQb_array_next,model->pQb_array_next, model->sizeof_pQb_array*model->rows*model->columns*sizeof(CALbyte), hipMemcpyHostToDevice);
	}
	if(model->sizeof_pQi_array > 0){
		hipMemcpy(copy_model->pQi_array_current,model->pQi_array_current, model->sizeof_pQi_array*model->rows*model->columns*sizeof(CALint), hipMemcpyHostToDevice);
		hipMemcpy(copy_model->pQi_array_next,model->pQi_array_next, model->sizeof_pQi_array*model->rows*model->columns*sizeof(CALint), hipMemcpyHostToDevice);
	}
	if(model->sizeof_pQr_array > 0){
		hipMemcpy(copy_model->pQr_array_current,model->pQr_array_current, model->sizeof_pQr_array*model->rows*model->columns*sizeof(CALreal), hipMemcpyHostToDevice);
		hipMemcpy(copy_model->pQr_array_next,model->pQr_array_next, model->sizeof_pQr_array*model->rows*model->columns*sizeof(CALreal), hipMemcpyHostToDevice);
	}
	hipMemcpy(d_model, copy_model, sizeof(struct CudaCALModel2D), hipMemcpyHostToDevice);

	return result;
}

CALbyte calSendDataGPUtoCPU(struct CudaCALModel2D* model, struct CudaCALModel2D *d_model){

	CALbyte result = CAL_TRUE;

	hipMemcpy(copy_model, d_model, sizeof(struct CudaCALModel2D), hipMemcpyDeviceToHost);

	if(model->sizeof_pQb_array > 0){
		hipMemcpy(model->pQb_array_current,copy_model->pQb_array_current,model->sizeof_pQb_array*model->rows*model->columns*sizeof(CALbyte), hipMemcpyDeviceToHost);
		hipMemcpy(model->pQb_array_next,copy_model->pQb_array_next,model->sizeof_pQb_array*model->rows*model->columns*sizeof(CALbyte), hipMemcpyDeviceToHost);		
	}
	if(model->sizeof_pQi_array > 0){
		hipMemcpy(model->pQi_array_current,copy_model->pQi_array_current,model->sizeof_pQi_array*model->rows*model->columns*sizeof(CALint), hipMemcpyDeviceToHost);		
		hipMemcpy(model->pQi_array_next,copy_model->pQi_array_next,model->sizeof_pQi_array*model->rows*model->columns*sizeof(CALint), hipMemcpyDeviceToHost);
	}
	if(model->sizeof_pQr_array > 0){
		hipMemcpy(model->pQr_array_current,copy_model->pQr_array_current,model->sizeof_pQr_array*model->rows*model->columns*sizeof(CALreal), hipMemcpyDeviceToHost);		
		hipMemcpy(model->pQr_array_next,copy_model->pQr_array_next,model->sizeof_pQr_array*model->rows*model->columns*sizeof(CALreal), hipMemcpyDeviceToHost);
	}

	calCudaFinalizeModel();

	return result;
}

void printError(hipError_t error){
	//if(error != hipSuccess)
	printf("Error: %s\n", error);
}

void cudaErrorCheck(char* message){	
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{	
		printf("\n******************\n*** Attention! *** \n******************\n"); 
		printf("\nError: %s \nName: %s\n", hipGetErrorString(error), hipGetErrorName(error));
		printf("\n******************\n\n"); 
		system("pause");
		exit(-1);
	}else{
		printf("Message: %s\n", message);
	}
}

void cudaErrorCheck(char* message, CALbyte &result){	
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{	
		printf("\n******************\n*** Attention! *** \n******************\n"); 
		printf("\nError: %s \nName: %s\n", hipGetErrorString(error), hipGetErrorName(error));
		printf("\n******************\n\n"); 
		result = CAL_FALSE;
		system("pause");
		exit(-1);
	}else{
		printf("Message: %s\n", message);
	}
}

// SCIDDICA_T TEST
#define OUTPUT_PATH "./data/width_final.txt"
#define OUTPUT_PATH_S "./data/width_final_s.txt"
CALbyte calCudaCheckFinalResult2Dr(CALreal* parallel, CALreal* sequential, CALint rows, CALint columns){

	CALint i,j;
	hipHostAlloc((void**)&parallel, sizeof(CALreal)*rows*columns, hipHostMallocDefault); 
	hipHostAlloc((void**)&sequential,	sizeof(CALreal)*rows*columns, hipHostMallocDefault);

	calCudaLoadMatrix2Dr(parallel, rows, columns, OUTPUT_PATH,0);
	calCudaLoadMatrix2Dr(sequential, rows, columns, OUTPUT_PATH_S,0);

	for(i=0; i<rows; i++)
		for(j=0; j<columns; j++)
		{
			if(parallel[i*columns+j] != sequential[i*columns+j])
				return CAL_FALSE;
		}

		return CAL_TRUE;
}

__device__ CALbyte calCudaImAlive(struct CudaCALModel2D* ca2D, CALint offset){
	return calCudaGetMatrixElement(ca2D->activecell_flags, offset, ca2D->rows, ca2D->columns, 0); 
}

void calCudaPerformGridAndBlockForStreamCompaction2D(struct CudaCALRun2D* simulation, dim3 grid, dim3 block){
	simulation->ca2D->activecell_size_current = simulation->h_device_ca2D->activecell_size_next;
	CALint num_blocks = simulation->ca2D->activecell_size_current / ((block.x) * (block.y));
	grid.x = (num_blocks+2);
	grid.y = 1;
}
